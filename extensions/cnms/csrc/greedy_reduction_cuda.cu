#include "hip/hip_runtime.h"
// greedy_reduction_cuda.cu
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void shared_memory_greedy_reduction_kernel(
    const int* __restrict__ sorted_indices,
    const int* __restrict__ idx,
    const int* __restrict__ lengths,
    bool* __restrict__ retain,
    int num_batches,
    int num_spheres,
    int num_neighbors,
    int ignore_idx
) {
    extern __shared__ bool shared_retain[];
    
    int batch_idx = blockIdx.x;
    int tid = threadIdx.x;
    
    if (batch_idx >= num_batches) return;
    
    int valid_length = lengths[batch_idx];
    
    // init shared memory
    for (int i = tid; i < num_spheres; i += blockDim.x) {
        shared_retain[i] = (i < valid_length);
    }
    __syncthreads();
    
    // process spheres in sorted order
    for (int i = 0; i < num_spheres; i++) {
        int sphere_idx = sorted_indices[batch_idx * num_spheres + i];
        
        if (!shared_retain[sphere_idx]) continue;
        
        // process neighbors
        for (int j = tid; j < num_neighbors; j += blockDim.x) {
            int neighbor = idx[batch_idx * num_spheres * num_neighbors + sphere_idx * num_neighbors + j];
            if (neighbor != sphere_idx && neighbor != ignore_idx) {
                shared_retain[neighbor] = false;
            }
        }
        __syncthreads();
    }
    
    // write back to global memory
    for (int i = tid; i < num_spheres; i += blockDim.x) {
        retain[batch_idx * num_spheres + i] = shared_retain[i];
    }
}


__global__ void optimized_greedy_reduction_kernel(
    const int* __restrict__ sorted_indices,
    const int* __restrict__ idx,
    const int* __restrict__ lengths,
    bool* __restrict__ retain,
    int num_batches,
    int num_spheres,
    int num_neighbors,
    int ignore_idx
) {
    int batch_idx = blockIdx.x;
    int sphere_offset = blockIdx.y * blockDim.x + threadIdx.x;
    
    if (batch_idx >= num_batches) return;
    
    int valid_length = lengths[batch_idx];
    
    // init retain array
    for (int i = sphere_offset; i < num_spheres; i += blockDim.x * gridDim.y) {
        retain[batch_idx * num_spheres + i] = (i < valid_length);
    }
    __syncthreads();
    
    // process spheres in sorted order
    for (int i = 0; i < num_spheres; i++) {
        int sphere_idx = sorted_indices[batch_idx * num_spheres + i];
        
        bool should_process = retain[batch_idx * num_spheres + sphere_idx];
        __syncthreads();
        
        if (!should_process) continue;
        
        // process neighbors
        for (int j = sphere_offset; j < num_neighbors; j += blockDim.x * gridDim.y) {
            int neighbor = idx[batch_idx * num_spheres * num_neighbors + sphere_idx * num_neighbors + j];
            if (neighbor != sphere_idx && neighbor != ignore_idx) {
                atomicAnd((int*)&retain[batch_idx * num_spheres + neighbor], 0);
            }
        }
        __syncthreads();
    }
}


void launch_greedy_reduction_cuda_kernel(
    const int* sorted_indices,
    const int* idx,
    const int* lengths,
    bool* retain,
    int num_batches,
    int num_spheres,
    int num_neighbors,
    int ignore_idx
) {
    // // Define CUDA grid and block dimensions
    // int threads = 256;
    // int blocks = (num_batches + threads - 1) / threads;

    // // Launch the CUDA kernel
    // greedy_reduction_cuda_kernel<<<blocks, threads>>>(
    //     sorted_indices,
    //     idx,
    //     lengths,
    //     retain,
    //     num_batches,
    //     num_spheres,
    //     num_neighbors,
    //     ignore_idx
    // );

    dim3 blocks(num_batches, min(32, num_spheres)); // up to 32 blocks in y dimension
    int threads = 256;
    
    // shared memory version
    int threads_shared = 256;
    int blocks_shared = num_batches;
    int shared_mem_size = num_spheres * sizeof(bool);
    
    if (num_spheres <= 4096) {  // use shared memory for small sizes
        shared_memory_greedy_reduction_kernel<<<blocks_shared, threads_shared, shared_mem_size>>>(
            sorted_indices, idx, lengths, retain, num_batches, num_spheres, num_neighbors, ignore_idx);
    } else {
        optimized_greedy_reduction_kernel<<<blocks, threads>>>(
            sorted_indices, idx, lengths, retain, num_batches, num_spheres, num_neighbors, ignore_idx);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Kernel Failed: %s\n", hipGetErrorString(err));
    }
}